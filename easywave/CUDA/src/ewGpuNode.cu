#include "hip/hip_runtime.h"
/*
 * EasyWave - A realtime tsunami simulation program with GPU support.
 * Copyright (C) 2014  Andrey Babeyko, Johannes Spazier
 * GFZ German Research Centre for Geosciences (http://www.gfz-potsdam.de)
 *
 * Parts of this program (especially the GPU extension) were developed
 * within the context of the following publicly funded project:
 * - TRIDEC, EU 7th Framework Programme, Grant Agreement 258723
 *   (http://www.tridec-online.eu)
 *
 * Licensed under the EUPL, Version 1.1 or - as soon they will be approved by
 * the European Commission - subsequent versions of the EUPL (the "Licence"),
 * complemented with the following provision: For the scientific transparency
 * and verification of results obtained and communicated to the public after
 * using a modified version of the work, You (as the recipient of the source
 * code and author of this modified version, used to produce the published
 * results in scientific communications) commit to make this modified source
 * code available in a repository that is easily and freely accessible for a
 * duration of five years after the communication of the obtained results.
 *
 * You may not use this work except in compliance with the Licence.
 *
 * You may obtain a copy of the Licence at:
 * https://joinup.ec.europa.eu/software/page/eupl
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the Licence is distributed on an "AS IS" basis,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the Licence for the specific language governing permissions and
 * limitations under the Licence.
 */

/*
 * Modifications Copyright (C) 2023 Intel Corporation
 * 
 * This Program is subject to the terms of the European Union Public License 1.2
 * 
 * If a copy of the license was not distributed with this file, you can obtain one at 
 * https://joinup.ec.europa.eu/sites/default/files/custom-page/attachment/2020-03/EUPL-1.2%20EN.txt
 * 
 * SPDX-License-Identifier: EUPL-1.2
 */

#define CPP_MODULE "CGPU"
#include "Logging.h"

#include "ewGpuNode.cuh"
#include "ewCudaKernels.cuh"
#include <algorithm>
#include <chrono>
#include <cassert>

CGpuNode::CGpuNode()
{

    pitch  = 0;
    copied = true;

    m_vecTimers.emplace_back(Timer("MemcpyD2H")); // 0
    m_vecTimers.emplace_back(Timer("MemcpyH2D")); // 1
    m_vecTimers.emplace_back(Timer("MemFree"));   // 2
    m_vecTimers.emplace_back(Timer("MemAlloc"));  // 3
    m_vecTimers.emplace_back(Timer("Compute"));   // 4

#ifdef ENABLE_KERNEL_PROFILING
    for (int i = 0; i < 5; i++) {
        hipEventCreate(&(evtStart[i]));
        hipEventCreate(&(evtEnd[i]));
        dur[i] = 0.0;
    }
#endif

    pMallocPitch_DoNotUse = nullptr;
}

void CGpuNode::PrintTimingStats()
{
#ifdef ENABLE_GPU_TIMINGS
    LOG("Timing Results | Elapsed time");
    LOG("*****************************");
    LOG("Memory Alloc    | " << m_vecTimers[TIMER_MEMALLOC].GetTimeAsString(Timer::Units::SECONDS));
    LOG("Memory Xfer H2D | " << m_vecTimers[TIMER_MEMH2D].GetTimeAsString(Timer::Units::SECONDS));
    LOG("Memory Xfer D2H | " << m_vecTimers[TIMER_MEMD2H].GetTimeAsString(Timer::Units::SECONDS));
    LOG("Memory free     | " << m_vecTimers[TIMER_MEMFREE].GetTimeAsString(Timer::Units::SECONDS));
    LOG("Compute         | " << m_vecTimers[TIMER_COMPUTE].GetTimeAsString(Timer::Units::SECONDS));
    LOG("Total GPU Time  | " << (m_vecTimers[TIMER_MEMALLOC] + m_vecTimers[TIMER_MEMH2D] + m_vecTimers[TIMER_MEMD2H] + m_vecTimers[TIMER_MEMFREE] + m_vecTimers[TIMER_COMPUTE]).GetTimeAsString(Timer::Units::SECONDS));
    LOG("*****************************");

#ifdef ENABLE_KERNEL_PROFILING
    // TODO: FIX KERNEL TIMERS!!!
    LOG("Kernel time(s)");
    LOG("\t Wave Update  : " << Utility::ConvertTimeToReadable(Node.GetKernelTimeInMilliseconds(KERNEL_WAVE_UPDATE)));
    LOG("\t Wave Boundary: " << Utility::ConvertTimeToReadable(Node.GetKernelTimeInMilliseconds(KERNEL_WAVE_BOUNDARY)));
    LOG("\t Flux Update  : " << Utility::ConvertTimeToReadable(Node.GetKernelTimeInMilliseconds(KERNEL_FLUX_UPDATE)));
    LOG("\t Flux Boundary: " << Utility::ConvertTimeToReadable(Node.GetKernelTimeInMilliseconds(KERNEL_FLUX_BOUNDARY)));
    LOG("\t Grid Extend  : " << Utility::ConvertTimeToReadable(Node.GetKernelTimeInMilliseconds(KERNEL_GRID_EXTEND)));
#endif
#endif
}

int CGpuNode::mallocMem()
{

    LOG("Allocating GPU memory");

    CArrayNode::mallocMem();

    Params &dp = data.params;

    /* fill in some fields here */
    dp.nI                  = NLon;
    dp.nJ                  = NLat;
    dp.sshArrivalThreshold = Par.sshArrivalThreshold;
    dp.sshClipThreshold    = Par.sshClipThreshold;
    dp.sshZeroThreshold    = Par.sshZeroThreshold;
    dp.lpad                = 0;

    size_t nJ_aligned = dp.nJ + dp.lpad;

    // hipMallocPitch is needed here in order to obtain the pitch value

#ifdef ENABLE_GPU_TIMINGS
    m_vecTimers[TIMER_MEMALLOC].Start();
#endif

    CUDA_CALL(hipMallocPitch(&(pMallocPitch_DoNotUse), &pitch, nJ_aligned * sizeof(float), dp.nI));
    LOG_ASSERT(pitch != 0, "Failed to compute pitch");
    LOG("Computed pitch in bytes is: " << pitch << ", dp.pI: " << pitch / sizeof(float));

    CUDA_CALL(hipMalloc(&(data.d), pitch * dp.nI));
    CUDA_CALL(hipMalloc(&(data.h), pitch * dp.nI));
    CUDA_CALL(hipMalloc(&(data.hMax), pitch * dp.nI));
    CUDA_CALL(hipMalloc(&(data.fM), pitch * dp.nI));
    CUDA_CALL(hipMalloc(&(data.fN), pitch * dp.nI));
    CUDA_CALL(hipMalloc(&(data.cR1), pitch * dp.nI));
    CUDA_CALL(hipMalloc(&(data.cR2), pitch * dp.nI));
    CUDA_CALL(hipMalloc(&(data.cR4), pitch * dp.nI));
    CUDA_CALL(hipMalloc(&(data.tArr), pitch * dp.nI));
    /* TODO: cR3, cR5 for coriolis */

    /* 1-dim */
    CUDA_CALL(hipMalloc(&(data.cR6), dp.nJ * sizeof(float)));
    CUDA_CALL(hipMalloc(&(data.cB1), dp.nI * sizeof(float)));
    CUDA_CALL(hipMalloc(&(data.cB2), dp.nJ * sizeof(float)));
    CUDA_CALL(hipMalloc(&(data.cB3), dp.nI * sizeof(float)));
    CUDA_CALL(hipMalloc(&(data.cB4), dp.nJ * sizeof(float)));

    CUDA_CALL(hipMalloc(&(data.g_MinMax), sizeof(int4)));

#ifdef ENABLE_GPU_TIMINGS
    m_vecTimers[TIMER_MEMALLOC].Stop();
#endif

    /* TODO: make sure that pitch is a multiple of 4 and the same for each hipMallocPitch() call */
    dp.pI = pitch / sizeof(float);

    return 0;
}

bool CGpuNode::AlignData(float const *const &pInputData, float *&pOutputAlignedData, int const iNumberOfRows,
                         int const iNumberOfCols,        // From Pitch
                         int const iNumberOfColsPitched) // To Pitch
{
    assert(pInputData != nullptr && pOutputAlignedData != nullptr);
    for (int iRow = 0; iRow < iNumberOfRows; ++iRow) {
        std::move(pInputData + (iRow * iNumberOfCols), pInputData + (iRow * iNumberOfCols) + iNumberOfCols, pOutputAlignedData + (iRow * iNumberOfColsPitched));
    }

    return true;
}

int CGpuNode::copyToGPU()
{

    Params &dp = data.params;

    /* align left grid boundary to a multiple of 32 with an offset 1 */
    Jmin -= (Jmin - 2) % 32;

    /* fill in further fields here */
    dp.iMin = Imin;
    dp.iMax = Imax;
    dp.jMin = Jmin;
    dp.jMax = Jmax;

    d_1D_aligned    = new float[dp.nI * dp.pI];
    h_1D_aligned    = new float[dp.nI * dp.pI];
    hMax_1D_aligned = new float[dp.nI * dp.pI];
    fM_1D_aligned   = new float[dp.nI * dp.pI];
    fN_1D_aligned   = new float[dp.nI * dp.pI];
    cR1_1D_aligned  = new float[dp.nI * dp.pI];
    cR2_1D_aligned  = new float[dp.nI * dp.pI];
    cR4_1D_aligned  = new float[dp.nI * dp.pI];
    tArr_1D_aligned = new float[dp.nI * dp.pI];

    assert(d_1D_aligned != nullptr);
    assert(h_1D_aligned != nullptr);
    assert(hMax_1D_aligned != nullptr);
    assert(fM_1D_aligned != nullptr);
    assert(fN_1D_aligned != nullptr);
    assert(cR1_1D_aligned != nullptr);
    assert(cR2_1D_aligned != nullptr);
    assert(cR4_1D_aligned != nullptr);
    assert(tArr_1D_aligned != nullptr);

    AlignData(d, d_1D_aligned, dp.nI, dp.nJ, dp.pI);
    AlignData(h, h_1D_aligned, dp.nI, dp.nJ, dp.pI);
    AlignData(hMax, hMax_1D_aligned, dp.nI, dp.nJ, dp.pI);
    AlignData(fM, fM_1D_aligned, dp.nI, dp.nJ, dp.pI);
    AlignData(fN, fN_1D_aligned, dp.nI, dp.nJ, dp.pI);
    AlignData(cR1, cR1_1D_aligned, dp.nI, dp.nJ, dp.pI);
    AlignData(cR2, cR2_1D_aligned, dp.nI, dp.nJ, dp.pI);
    AlignData(cR4, cR4_1D_aligned, dp.nI, dp.nJ, dp.pI);
    AlignData(tArr, tArr_1D_aligned, dp.nI, dp.nJ, dp.pI);

#ifdef ENABLE_GPU_TIMINGS
    m_vecTimers[TIMER_MEMH2D].Start();
#endif

    /* add offset to data.d to guarantee alignment: data.d + LPAD */
    /* 2-dim */

    CUDA_CALL(hipMemcpy(data.d, d_1D_aligned, dp.nI * dp.pI * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(data.h, h_1D_aligned, dp.nI * dp.pI * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(data.hMax, hMax_1D_aligned, dp.nI * dp.pI * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(data.fM, fM_1D_aligned, dp.nI * dp.pI * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(data.fN, fN_1D_aligned, dp.nI * dp.pI * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(data.cR1, cR1_1D_aligned, dp.nI * dp.pI * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(data.cR2, cR2_1D_aligned, dp.nI * dp.pI * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(data.cR4, cR4_1D_aligned, dp.nI * dp.pI * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(data.tArr, tArr_1D_aligned, dp.nI * dp.pI * sizeof(float), hipMemcpyHostToDevice));

    /* FIXME: move global variables into data structure */
    /* 1-dim */
    CUDA_CALL(hipMemcpy(data.cR6, R6, dp.nJ * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(data.cB1, C1, dp.nI * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(data.cB2, C2, dp.nJ * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(data.cB3, C3, dp.nI * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(data.cB4, C4, dp.nJ * sizeof(float), hipMemcpyHostToDevice));

#ifdef ENABLE_GPU_TIMINGS
    m_vecTimers[TIMER_MEMH2D].Stop();

    LOG("Data copy to GPU completed, Time elapsed: " << m_vecTimers[TIMER_MEMH2D].GetTimeAsString(Timer::Units::SECONDS));
#endif

    return 0;
}
int CGpuNode::copyFromGPU()
{

    Params &dp = data.params;
#ifdef ENABLE_GPU_TIMINGS
    std::chrono::steady_clock::time_point const tStart(std::chrono::steady_clock::now());
#endif
    CUDA_CALL(hipMemcpy(hMax_1D_aligned, data.hMax, dp.nI * dp.pI * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(tArr_1D_aligned, data.tArr, dp.nI * dp.pI * sizeof(float), hipMemcpyDeviceToHost));
#ifdef ENABLE_GPU_TIMINGS
    std::chrono::steady_clock::time_point const tStop(std::chrono::steady_clock::now());
    m_vecTimers[TIMER_MEMD2H] += std::chrono::steady_clock::duration(tStop - tStart);
#endif
    AlignData(hMax_1D_aligned, hMax, dp.nI, dp.pI, dp.nJ);
    AlignData(tArr_1D_aligned, tArr, dp.nI, dp.pI, dp.nJ);

    return 0;
}

int CGpuNode::copyIntermediate()
{

    /* ignore copy requests if data already present on CPU side */
    if (copied)
        return 0;

    Params &dp = data.params;
#ifdef ENABLE_GPU_TIMINGS
    std::chrono::steady_clock::time_point const tStart(std::chrono::steady_clock::now());
#endif
    CUDA_CALL(hipMemcpy(h_1D_aligned, data.h, dp.nI * dp.pI * sizeof(float), hipMemcpyDeviceToHost));
#ifdef ENABLE_GPU_TIMINGS
    std::chrono::steady_clock::time_point const tStop(std::chrono::steady_clock::now());
    m_vecTimers[TIMER_MEMD2H] += std::chrono::steady_clock::duration(tStop - tStart);
#endif
    AlignData(h_1D_aligned, h, dp.nI, dp.pI, dp.nJ);
    /* copy finished */
    copied = true;

    return 0;
}

int CGpuNode::copyPOIs()
{

    LOG_WARNING("Copying POIs, this may prolong total time");
    Params &dp = data.params;

    if (copied)
        return 0;

    for (int n = 0; n < NPOIs; n++) {

        int i = idxPOI[n] / dp.nJ + 1;
        int j = idxPOI[n] % dp.nJ + 1;

        int id = data.idx(i, j);
#ifdef ENABLE_GPU_TIMINGS
        std::chrono::steady_clock::time_point const tStart(std::chrono::steady_clock::now());
#endif
        CUDA_CALL(hipMemcpy(h + idxPOI[n], data.h + id, sizeof(float), hipMemcpyDeviceToHost));
#ifdef ENABLE_GPU_TIMINGS
        std::chrono::steady_clock::time_point const tStop(std::chrono::steady_clock::now());
        m_vecTimers[TIMER_MEMD2H] += std::chrono::steady_clock::duration(tStop - tStart);
#endif
    }

    return 0;
}

int CGpuNode::freeMem()
{

    /* 2-dim */
#ifdef ENABLE_GPU_TIMINGS
    m_vecTimers[TIMER_MEMFREE].Start();
#endif
    CUDA_CALL(hipFree(data.d));
    CUDA_CALL(hipFree(data.h));
    CUDA_CALL(hipFree(data.hMax));
    CUDA_CALL(hipFree(data.fM));
    CUDA_CALL(hipFree(data.fN));
    CUDA_CALL(hipFree(data.cR1));
    CUDA_CALL(hipFree(data.cR2));
    CUDA_CALL(hipFree(data.cR4));
    CUDA_CALL(hipFree(data.tArr));

    /* 1-dim */
    CUDA_CALL(hipFree(data.cR6));
    CUDA_CALL(hipFree(data.cB1));
    CUDA_CALL(hipFree(data.cB2));
    CUDA_CALL(hipFree(data.cB3));
    CUDA_CALL(hipFree(data.cB4));

    CUDA_CALL(hipFree(data.g_MinMax));
    CUDA_CALL(hipFree(pMallocPitch_DoNotUse));
#ifdef ENABLE_GPU_TIMINGS
    m_vecTimers[TIMER_MEMFREE].Stop();
#endif

#ifdef ENABLE_KERNEL_PROFILING
    float total_dur = 0.f;
    for (int j = 0; j < 5; j++) {
        printf_v("Duration %u: %.3f\n", j, dur[j]);
        total_dur += dur[j];
    }
    printf_v("Duration total: %.3f\n", total_dur);
#endif

    CArrayNode::freeMem();

    return 0;
}

int CGpuNode::run()
{

    Params &dp = data.params;

    int nThreads = 256;
    int xThreads = 32;
    int yThreads = nThreads / xThreads;

    int NJ      = dp.jMax - dp.jMin + 1;
    int NI      = dp.iMax - dp.iMin + 1;
    int xBlocks = ceil((float)NJ / (float)xThreads);
    int yBlocks = ceil((float)NI / (float)yThreads);

    dim3 threads(xThreads, yThreads);
    dim3 blocks(xBlocks, yBlocks);

    int nBlocks = ceil((float)std::max(dp.nI, dp.nJ) / (float)nThreads);

    dp.mTime = Par.time;

#ifdef ENABLE_KERNEL_PROFILING
    CUDA_CALL(hipEventRecord(evtStart[0], 0));
    runWaveUpdateKernel<<<blocks, threads>>>(data);
    CUDA_CALL(hipEventRecord(evtEnd[0], 0));
    CUDA_CALL(hipEventRecord(evtStart[1], 0));
    runWaveBoundaryKernel<<<nBlocks, nThreads>>>(data);
    CUDA_CALL(hipEventRecord(evtEnd[1], 0));
    CUDA_CALL(hipEventRecord(evtStart[2], 0));
    runFluxUpdateKernel<<<blocks, threads>>>(data);
    CUDA_CALL(hipEventRecord(evtEnd[2], 0));
    CUDA_CALL(hipEventRecord(evtStart[3], 0));
    runFluxBoundaryKernel<<<nBlocks, nThreads>>>(data);
    CUDA_CALL(hipEventRecord(evtEnd[3], 0));
    CUDA_CALL(hipEventRecord(evtStart[4], 0));
    CUDA_CALL(hipMemset(data.g_MinMax, 0, sizeof(int4)));
    runGridExtendKernel<<<nBlocks, nThreads>>>(data);
    CUDA_CALL(hipEventRecord(evtEnd[4], 0));
#else
#ifdef ENABLE_GPU_TIMINGS
    std::chrono::steady_clock::time_point const tStart(std::chrono::steady_clock::now());
#endif
    runWaveUpdateKernel<<<blocks, threads>>>(data);
    checkLastCUDAError();
    runWaveBoundaryKernel<<<nBlocks, nThreads>>>(data);
    checkLastCUDAError();
    runFluxUpdateKernel<<<blocks, threads>>>(data);
    checkLastCUDAError();
    runFluxBoundaryKernel<<<nBlocks, nThreads>>>(data);
    checkLastCUDAError();
    CUDA_CALL(hipMemset(data.g_MinMax, 0, sizeof(int4)));
    runGridExtendKernel<<<nBlocks, nThreads>>>(data);
    checkLastCUDAError();
    hipDeviceSynchronize();
    checkLastCUDAError();

#ifdef ENABLE_GPU_TIMINGS
    std::chrono::steady_clock::time_point const tStop(std::chrono::steady_clock::now());

    m_vecTimers[TIMER_COMPUTE] += std::chrono::steady_clock::duration(tStop - tStart);
#endif

#endif

    int4 MinMax;
#ifdef ENABLE_GPU_TIMINGS
    std::chrono::steady_clock::time_point const tStartD2H(std::chrono::steady_clock::now());
#endif
    CUDA_CALL(hipMemcpy(&MinMax, data.g_MinMax, sizeof(int4), hipMemcpyDeviceToHost));
#ifdef ENABLE_GPU_TIMINGS
    std::chrono::steady_clock::time_point const tStopD2H(std::chrono::steady_clock::now());
    m_vecTimers[TIMER_MEMD2H] += std::chrono::duration(tStopD2H - tStartD2H);
#endif

    if (MinMax.x)
        Imin = dp.iMin = std::max(dp.iMin - 1, 2);

    if (MinMax.y)
        Imax = dp.iMax = std::min(dp.iMax + 1, dp.nI - 1);

    if (MinMax.z)
        Jmin = dp.jMin = std::max(dp.jMin - 32, 2);

    if (MinMax.w)
        Jmax = dp.jMax = std::min(dp.jMax + 1, dp.nJ - 1);

#ifdef ENABLE_KERNEL_PROFILING
    float _dur;
    for (int j = 0; j < 5; j++) {
        hipEventElapsedTime(&_dur, evtStart[j], evtEnd[j]);
        dur[j] += _dur;
    }
#endif

    /* data has changed now -> copy becomes necessary */
    copied = false;

    return 0;
}
